#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <time.h>

/**
* Michael O'Brien 20241223
* michael at obrienlabs.dev
* 128 bit version
* Collatz sequence running on NVidia GPUs like the RTX-3500 ada,A4000,A4500,4090 ada and A6000
* http://www.ericr.nl/wondrous/pathrecs.html
* https://github.com/ObrienlabsDev/performance
* https://github.com/obrienlabs/benchmark/blob/master/ObjectiveC/128bit/main.m
* https://github.com/obrienlabs/benchmark/blob/master/collatz_vs10/collatz_vs10/collatz_vs10.cpp
* https://github.com/ObrienlabsDev/cuda/blob/main/add_example/kernel_collatz.cu
* https://github.com/ObrienlabsDev/collatz/blob/main/src/main/java/dev/obrienlabs/collatz/service/CollatzUnitOfWork.java
* 
*/


/* CUDA Kernel runs on GPU device streaming core */
__global__ void collatzCUDAKernel(unsigned long long* _input, unsigned long long* _output, int threads)//, unsigned long long iterations)
{
    // Calculate this thread's index
    int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Check boundary (in case N is not a multiple of blockDim.x)
    int path = 0;
    unsigned long long max = _input[threadIndex];
    unsigned long long current = _input[threadIndex];

    if (threadIndex < threads)
    {
            path = 0;
            max = _input[threadIndex];
            current = _input[threadIndex];

            do {
                path += 1;
                if (current % 2 == 0) {
                    current = current >> 1;
                }
                else {
                    current = 1 + current * 3;
                    if (current > max) {
                        max = current;
                    }
                }
            } while (current > 1);
    }
    _output[threadIndex] = max;
}

void singleGPUSearch() {
    int deviceCount = 0;
    int dualDevice = 0;
    hipGetDeviceCount(&deviceCount);
    printf("%d CUDA devices found - reallocating\n", deviceCount);
    if (deviceCount > 1) {
        dualDevice = 1;
    }

    const int dev0 = 0;
    const int dev1 = 1;

    int cores = 5120;// (argc > 1) ? atoi(argv[1]) : 5120; // get command
    // exited with code -1073741571 any higher
    // VRAM related - cannot exceed 32k threads for dual 12g RTX-3500 - check 4090
    const int threads = 32768;// 32768;// 7168 * 7;// 7168 * 4;// 32768 - (1536);// 32768 - 1536;// 7168 * 4;
    // 22sec on 7168 * 6 = 43008 55% gpu
    // 21-24 sec on 7168*7
    // 24 sec on 7168 * 8
    // 32 sec on 16384

    const int threadsPerBlock = 256;// 128;// 128; 128=50%, 256=66 on RTX-3500
    // Host arrays
    unsigned long long host_input0[threads];
    
    unsigned long long startSequence = 1L;
    unsigned long long globalMaxValue = 1L;
    unsigned long long globalMaxStart = startSequence;
    unsigned long long endSequence = 1 << 16; // 20 = 190 sec
    unsigned long long batchNumber = (endSequence - startSequence + 1) ;
    printf("%d\n", batchNumber);

    unsigned long long host_result0[threads] = { 0 };
    unsigned long long* device_input0 = nullptr;
    unsigned long long* device_output0 = nullptr;

    time_t timeStart, timeEnd;
    double timeElapsed;

    time(&timeStart);

    // Allocate memory on the GPU
    size_t size = threads * sizeof(unsigned long long);
    printf("array allocation bytes per GPU: %d * %d is %d maxSearch: %lld\n", sizeof(unsigned long long), threads, size, startSequence);
    // Number of blocks = ceiling(N / threadsPerBlock)
    int blocks = (threads + threadsPerBlock - 1) / threadsPerBlock;

    hipSetDevice(dev0);
    hipMalloc((void**)&device_input0, size);
    hipMalloc((void**)&device_output0, size);

    printf("GPU0: Threads: %d ThreadsPerBlock: %d Blocks: %d\n", threads, threadsPerBlock, blocks);
    for (int batch = 0; batch < endSequence; batch++) {
        // prepare inputs
        for (int thread = 0; thread < threads; thread++) {
            host_input0[thread] = startSequence;
            startSequence += 2;
        }

        hipSetDevice(dev0);
        hipMemcpy(device_input0, host_input0, size, hipMemcpyHostToDevice);
        // Launch kernel
        hipSetDevice(dev0);
        // kernelName<<<numBlocks, threadsPerBlock>>>(parameters...);
        collatzCUDAKernel << <blocks, threadsPerBlock >> > (device_input0, device_output0, threads);

        // Wait for GPU to finish before accessing on host
        hipSetDevice(dev0);
        hipDeviceSynchronize();

        // Copy result from device back to host
        hipMemcpy(host_result0, device_output0, size, hipMemcpyDeviceToHost);

        // process reesults: parallelize with OpenMP
        for (int thread = 0; thread < threads; thread++)
        {
            if (host_result0[thread] > globalMaxValue) {
                globalMaxValue = host_result0[thread];
                globalMaxStart = host_input0[thread];
                time(&timeEnd);
                timeElapsed = difftime(timeEnd, timeStart);
                std::cout << "GPU0:Sec: " << timeElapsed << " GlobalMax: " << globalMaxStart << ": " << globalMaxValue << " last search: " << startSequence << "\n";
            }
        }
    }

    // Print the result for the last run
    std::cout << "collatz:\n";
    int i = 0;
    for (int i = 0; i < 20/*threads*/; i++)
    {
        std::cout << "GPU0: " << i << ": " << host_input0[i] << " = " << host_result0[i] << "\n";
    }

    time(&timeEnd);
    timeElapsed = difftime(timeEnd, timeStart);

    printf("duration: %.f\n", timeElapsed);
    std::cout << "Sec: " << timeElapsed << " GlobalMax: " << globalMaxStart << " : " << globalMaxValue << " last search : " << startSequence << "\n";

    // Free GPU memory
    hipSetDevice(dev0);
    hipFree(device_input0);
    hipFree(device_output0);

    free(host_input0);
    return;
}

void dualGPUSearch() {
    int deviceCount = 0;
    int dualDevice = 0;
    hipGetDeviceCount(&deviceCount);
    printf("%d CUDA devices found - reallocating", deviceCount);
    if (deviceCount > 1) {
        dualDevice = 1;
    }

    const int dev0 = 0;
    const int dev1 = 1;

    int cores = 5120;// (argc > 1) ? atoi(argv[1]) : 5120; // get command
    // exited with code -1073741571 any higher
    // VRAM related - cannot exceed 32k threads for dual 12g RTX-3500 - check 4090
    const int threads = 16384;// 32768;// 7168 * 7;// 7168 * 4;// 32768 - (1536);// 32768 - 1536;// 7168 * 4;
    // 22sec on 7168 * 6 = 43008 55% gpu
    // 21-24 sec on 7168*7
    // 24 sec on 7168 * 8
    // 32 sec on 16384

    const int threadsPerBlock = 256;// 128; 128=50%, 256=66 on RTX-3500
    // Host arrays
    unsigned long long host_input0[threads];
    unsigned long long host_input1[threads];

    unsigned long long startSequence = 1L;
    unsigned long long globalMaxValue = 1L;
    unsigned long long globalMaxStart = startSequence;
    unsigned long long endSequence = 1 << 16; // 20 = 190 sec

    unsigned long long host_result0[threads] = { 0 };
    unsigned long long host_result1[threads] = { 0 };

    // Device pointers
    unsigned long long* device_input0 = nullptr;
    unsigned long long* device_output0 = nullptr;
    unsigned long long* device_input1 = nullptr;
    unsigned long long* device_output1 = nullptr;

    time_t timeStart, timeEnd;
    double timeElapsed;

    time(&timeStart);

    // Allocate memory on the GPU
    size_t size = threads * sizeof(unsigned long long);
    printf("array allocation bytes per GPU: %d * %d is %d maxSearch: %lld\n", sizeof(unsigned long long), threads, size, startSequence);
    // Number of blocks = ceiling(N / threadsPerBlock)
    int blocks = (threads + threadsPerBlock - 1) / threadsPerBlock;

    hipSetDevice(dev0);
    hipMalloc((void**)&device_input0, size);
    hipMalloc((void**)&device_output0, size);

    if (dualDevice > 0) {
        hipSetDevice(dev1);
        hipMalloc((void**)&device_input1, size);
        hipMalloc((void**)&device_output1, size);
    }

    // prep for iteration
    int x;
    // 32k - 1.5k
    // GPU0: Iterations: 8388608 Threads: 31232 ThreadsPerBlock: 64 Blocks: 488
    printf("GPU0: Threads: %d ThreadsPerBlock: %d Blocks: %d\n", threads, threadsPerBlock, blocks);
    if (dualDevice > 0) {
        printf("GPU1: Threads: %d ThreadsPerBlock: %d Blocks: %d\n", threads, threadsPerBlock, blocks);
    }

    for (x = 0; x < endSequence; x++) {
        for (int q = 0; q < threads; q++) {
            host_input0[q] = startSequence;
            if (dualDevice > 0) {
                startSequence += 2;
                host_input1[q] = startSequence;
            }
            startSequence += 2;
        }

        hipSetDevice(dev0);
        hipMemcpy(device_input0, host_input0, size, hipMemcpyHostToDevice);

        if (dualDevice > 0) {
            hipSetDevice(dev1);
            hipMemcpy(device_input1, host_input1, size, hipMemcpyHostToDevice);
        }

        // Launch kernel
        hipSetDevice(dev0);
        // kernelName<<<numBlocks, threadsPerBlock>>>(parameters...);
        collatzCUDAKernel << <blocks, threadsPerBlock >> > (device_input0, device_output0, threads);

        if (dualDevice > 0) {
            hipSetDevice(dev1);
            collatzCUDAKernel << <blocks, threadsPerBlock >> > (device_input1, device_output1, threads);
        }

        // Wait for GPU to finish before accessing on host
        hipSetDevice(dev0);
        hipDeviceSynchronize();
        if (dualDevice > 0) {
            hipSetDevice(dev1);
            hipDeviceSynchronize();
        }

        // Copy result from device back to host
        hipMemcpy(host_result0, device_output0, size, hipMemcpyDeviceToHost);
        if (dualDevice > 0) {
            hipMemcpy(host_result1, device_output1, size, hipMemcpyDeviceToHost);
        }

        // parallelize
        for (int i = 0; i < threads; i++)
        {
            if (host_result0[i] > globalMaxValue) {
                globalMaxValue = host_result0[i];
                globalMaxStart = host_input0[i];
                time(&timeEnd);
                timeElapsed = difftime(timeEnd, timeStart);
                std::cout << "GPU0:Sec: " << timeElapsed << " GlobalMax: " << globalMaxStart << ": " << globalMaxValue << " last search: " << startSequence << "\n";
            }
            if (dualDevice > 0) {
                if (host_result1[i] > globalMaxValue) {
                    globalMaxValue = host_result1[i];
                    globalMaxStart = host_input1[i];
                    time(&timeEnd);
                    timeElapsed = difftime(timeEnd, timeStart);
                    std::cout << "GPU1:Sec: " << timeElapsed << " GlobalMax: " << globalMaxStart << ": " << globalMaxValue << " last search: " << startSequence << "\n";
                }
            }
        }
    }

    // Print the result
    std::cout << "collatz:\n";
    int i = 0;
    for (int i = 0; i < 20/*threads*/; i++)
    {
        std::cout << "GPU0: " << i << ": " << host_input0[i] << " = " << host_result0[i] << "\n";
        if (dualDevice > 0) {
            std::cout << "GPU1: " << i << ": " << host_input1[i] << " = " << host_result1[i] << "\n";
        }
    }

    time(&timeEnd);
    timeElapsed = difftime(timeEnd, timeStart);

    printf("duration: %.f\n", timeElapsed);
    std::cout << "Sec: " << timeElapsed << " GlobalMax: " << globalMaxStart << " : " << globalMaxValue << " last search : " << startSequence << "\n";

    // Free GPU memory
    hipSetDevice(dev0);
    hipFree(device_input0);
    hipFree(device_output0);
    if (dualDevice > 0) {
        hipSetDevice(dev1);
        hipFree(device_input1);
        hipFree(device_output1);
    }

    free(host_input0);
    if (dualDevice > 0) {
        free(host_input1);
    }
    return;
}

/* Host progrem */
int main(int argc, char* argv[])
{
    int cores = (argc > 1) ? atoi(argv[1]) : 5120; // get command
    singleGPUSearch();
    //dualGPUSearch();
    return 0;
}


#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <time.h>
#include <omp.h>

/**
* Michael O'Brien 20241223
* michael at obrienlabs.dev
* 128 bit version
* Collatz sequence running on NVidia GPUs like the RTX-3500 ada,A4000,A4500,4090 ada and A6000
* http://www.ericr.nl/wondrous/pathrecs.html
* https://github.com/ObrienlabsDev/performance
* https://github.com/obrienlabs/benchmark/blob/master/ObjectiveC/128bit/main.m
* https://github.com/obrienlabs/benchmark/blob/master/collatz_vs10/collatz_vs10/collatz_vs10.cpp
* https://github.com/ObrienlabsDev/cuda/blob/main/add_example/kernel_collatz.cu
* https://github.com/ObrienlabsDev/collatz/blob/main/src/main/java/dev/obrienlabs/collatz/service/CollatzUnitOfWork.java
*/

__global__ void collatzCUDAKernel(/*unsigned long long* _input1, */ unsigned long long* _input0,
    unsigned long long* _output1, unsigned long long* _output0, unsigned int* _path, int threads)
{
    const unsigned long long MAXBIT = 9223372036854775808ULL;
    const unsigned long long MAX64 = 18446744073709551615ULL;
    // Calculate this thread's index
    int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Check boundary (in case N is not a multiple of blockDim.x)
    unsigned int path = 0;
    unsigned long long max0 = 0ULL;
    unsigned long long current0 = 0ULL;
    unsigned long long max1 = 0ULL;
    unsigned long long current1 = 0ULL;
    unsigned long long temp0_shift = 0ULL;
    unsigned long long temp0_add = 0ULL;

    if (threadIndex < threads) {
            max0 = _input0[threadIndex];
            current0 = _input0[threadIndex];
            do {
                //_path[threadIndex] += 1;
                path += 1;
                // both even odd include a shift right - but 128 bit 2 bit carry math is required for large numbers at the 64 bit boundary
                if (current0 % 2ULL == 0) { // even
                    current0 = current0 >> 1;
                    // shift high byte if not odd (we already have a 0 in the MSB of the low word - no overflow will occur
                    if (current1 % 2ULL != 0) {
                        // add carry to avoid - overflow during the msb add to the low word
                        current0 += MAXBIT; // check overflow - will be none
                    }
                    current1 = current1 >> 1;
                } else { // odd
                    // odd n << 1 + n + 1
                    // use combined odd/even (n >> 1) + ceil(n) + 1 - only if 128 2 bit carry handling between
                    // do only 128-64 bit 3n part of 3n+1 (don't worry about overflow past 128bit into 256 bit space until we get past 64 bit inputs)
                    current1 *= 3ULL; // HIGH (3N)
                
                    // LOW (3N + 1) with 2 bit overflow
                    temp0_shift = (current0 << 1) + 1ULL; // shift first without bit0 carry in (do add n later)
                    // if lt - we have overflow
                    if (!(current0 < MAXBIT)) {//temp0_shift < current0
                        current1 += 1ULL; // add overflow carry
                    }

                    // add n step for odd - separate to break out possible 2 bit 64 bit boundary overflow
                    temp0_add = temp0_shift + current0;
                    if (temp0_add < current0) { // check shift left along with +1 instead of 
                        current1 += 1ULL; // add overflow carry
                    }

                    current0 = temp0_add;
                    // check for max (if combined odd/even mult by 2)
                    if (max1 < current1) {
                        max1 = current1;
                        max0 = current0;
                    } else {
                        if (max1 == current1) {
                            if (max0 < current0) {
                                max0 = current0;
                            }
                        }
                    }
                }
            } while (!((current0 == 1ULL) && (current1 == 0ULL)));
            // #31 move max copy inside the thread if check (to avoid concurrency issues)
            _output0[threadIndex] = max0;
            _output1[threadIndex] = max1;
            _path[threadIndex] = path;
    }
}

void singleGPUSearch() {
    unsigned long long MAXBIT = 9223372036854775808;
    unsigned int path = 0;
    int deviceCount = 0;
    int dualDevice = 0;
    hipGetDeviceCount(&deviceCount);
    printf("%d CUDA devices found - reallocating\n", deviceCount);
    if (deviceCount > 1) {
        dualDevice = 1;
    }

    const unsigned long long oddOffsetOptimization = 2ULL;
    const int dev0 = 0;
    const unsigned long long threadsPerBlock = 512ULL;// 128;// 128; 128=50%, 256=66 on RTX-3500
    unsigned long long cores = 5120ULL;// (argc > 1) ? atoi(argv[1]) : 5120; // get command

    // variables
    // keep these 2 in sync
    unsigned int threadsPower = 16;//20;// 16; // 15
    const unsigned long long threads = 7168 * 4 + 6144;// 40960;// 7168 * 2;// 40960;// 7168 * 5;// 32768; // maximize threads below 64k
    // 43008 crash rtx-3500
    // diff should be 31 bits (minus oddOffsetOptimization)
    unsigned int startSequencePower = 39;  // do not use 0
    unsigned int endSequencePower = 64; 

    // derived
    unsigned long long startSequenceNumber = (1ULL << startSequencePower) + 1ULL;
    unsigned long long endSequenceNumber = (1ULL << endSequencePower) - 1ULL;
    printf("endSequenceNumber: %llu\n", endSequenceNumber);
    // Number of blocks = ceiling(N / threadsPerBlock)
    unsigned int blocks = 1 * ((threads / threadsPerBlock));// +threadsPerBlock - 1) / threadsPerBlock);
    size_t size = threads * sizeof(unsigned long long);
    size_t sizeInt = threads * sizeof(unsigned int);
    unsigned long long globalMaxValue0 = startSequenceNumber;
    unsigned long long globalMaxStart0 = startSequenceNumber;
    unsigned long long globalMaxValue1 = 0ULL;
    unsigned long long globalMaxStart1 = 0ULL;
    unsigned long long iterations = (endSequenceNumber - startSequenceNumber) / oddOffsetOptimization;// *((1ULL << (endSequencePower - 32)));
    unsigned long long batchNumberPower = (endSequencePower - startSequencePower) - threadsPower;
    unsigned long long batchNumber = iterations / threads; // 1ULL << batchNumberPower;
    printf("BatchNumberPower: %llu\n", batchNumberPower);
    printf("BatchNumber: %llu\n", batchNumber);
    printf("Iterations: %llu\n", iterations);

    // Host arrays
    unsigned long long host_input0[threads];
    //unsigned long long host_input1[threads];
    unsigned long long host_result0[threads] = { 0ULL };
    unsigned long long* device_input0 = nullptr;
    unsigned long long* device_output0 = nullptr;
    // for 128 not 2nd GPU
    //unsigned long long* device_input1 = nullptr;
    unsigned long long* device_output1 = nullptr;
    unsigned int* device_path = nullptr;
    unsigned int host_path[threads] = { 0 };
    unsigned long long host_result1[threads] = { 0ULL };

    time_t timeStart, timeEnd;
    double timeElapsed;
    time(&timeStart);

    // Allocate memory on the GPU
    printf("array allocation bytes per GPU: %d * %d is %d maxSearch: %llu to %llu\n", sizeof(unsigned long long) * 2, threads, size, startSequenceNumber, endSequenceNumber);
    hipSetDevice(dev0);
    hipMalloc((void**)&device_input0, size);
    //hipMalloc((void**)&device_input1, size);
    hipMalloc((void**)&device_output0, size);
    hipMalloc((void**)&device_output1, size);
    hipMalloc((void**)&device_path, sizeInt);

    // Iterations = 2 ^ (15(threads) + 16(endSequence = runs) + 1(odd multiplier))
    printf("GPU0: Iterations: %llu via (Threads: %llu * Batches: %llu * 2 (odd mult)) ThreadsPerBlock: %d Blocks: %d\n", 
        iterations, threads, batchNumber, threadsPerBlock, blocks);
    for (int batch = 0; batch < batchNumber; batch++) {
        // prepare inputs
        for (int thread = 0; thread < threads; thread++) {
            host_input0[thread] = startSequenceNumber;
            //host_input1[thread] = 0ULL;
            startSequenceNumber += oddOffsetOptimization;
            host_path[thread] = 0;
        }

        hipMemcpy(device_input0, host_input0, size, hipMemcpyHostToDevice);
        //hipMemcpy(device_input1, host_input1, size, hipMemcpyHostToDevice);
        // Launch kernel
        // kernelName<<<numBlocks, threadsPerBlock>>>(parameters...);
        collatzCUDAKernel << <blocks, threadsPerBlock >> > (/*device_input1,*/ device_input0, device_output1, device_output0, device_path, threads);

        // Wait for GPU to finish before accessing on host
        hipDeviceSynchronize();

        // Copy result from device back to host
        hipMemcpy(host_result0, device_output0, size, hipMemcpyDeviceToHost);
        hipMemcpy(host_result1, device_output1, size, hipMemcpyDeviceToHost);
        hipMemcpy(host_path, device_path, sizeInt, hipMemcpyDeviceToHost);
        // process reesults: parallelize with OpenMP // no effect yet
        omp_set_num_threads(threads);
        #pragma omp parallel for reduction (+:globalMaxValue0, globalMaxValue1)
            for (int thread = 0; thread < threads; thread++) {
                path = host_path[thread];
                if (host_result1[thread] > globalMaxValue1) {
//#pragma omp critical
                   // {
                        globalMaxValue0 = host_result0[thread];
                        globalMaxValue1 = host_result1[thread];
                        globalMaxStart0 = host_input0[thread];
                        globalMaxStart1 = 0ULL;// host_input1[thread];

                        time(&timeEnd);
                        timeElapsed = difftime(timeEnd, timeStart);
                        std::cout << "GPU01:Sec: " << timeElapsed << " path: " << path << " GlobalMax: " << globalMaxStart1 << ":" << globalMaxStart0 << ": " << globalMaxValue1
                            << ":" << globalMaxValue0 << " last search: " << startSequenceNumber << "\n";
                    //}
                }
                else {
                    // handle only lsb gt
                    if (host_result1[thread] == globalMaxValue1) {
                        if (host_result0[thread] > globalMaxValue0) {
//#pragma omp critical 
                            //{
                                globalMaxValue0 = host_result0[thread];
                                globalMaxStart0 = host_input0[thread];
                                globalMaxStart1 = 0ULL;// host_input1[thread];

                                time(&timeEnd);
                                timeElapsed = difftime(timeEnd, timeStart);
                                std::cout << "GPU00:Sec: " << timeElapsed << " path: " << path << " GlobalMax: " << globalMaxStart1 << ":" << globalMaxStart0 << " : " << globalMaxValue1
                                    << ":" << globalMaxValue0 << " last search: " << startSequenceNumber << "\n";
                            //}
                        }
                    }
                }

            // TODO: maxPath
        }
    }

    // Print the result for the last run
    std::cout << "collatz:\n";
    for (int i = 0; i < 20/*threads*/; i++)
    {
        std::cout << "GPU0: " << i << ": " << /*host_input1[i] <<*/ ":" << host_input0[i] << " = " << host_result1[i] << host_result0[i] << "\n";
    }

    time(&timeEnd);
    timeElapsed = difftime(timeEnd, timeStart);

    printf("duration: %.f\n", timeElapsed);
    std::cout << "Sec: " << timeElapsed << " GlobalMax: " << globalMaxStart1 << ":" << globalMaxStart0 << " : " << globalMaxValue1 
        << ":" << globalMaxValue0 << " last search : " << startSequenceNumber << "\n";

    // Free GPU memory
    hipFree(device_input0);
    //hipFree(device_input1);
    hipFree(device_output0);
    hipFree(device_output1);
    hipFree(device_path);

    free(host_input0);
    //free(host_input1);
    free(host_result0);
    free(host_result1);
    free(host_path);
    return;
}

void dualGPUSearch() {
    int deviceCount = 0;
    int dualDevice = 0;
    hipGetDeviceCount(&deviceCount);
    printf("%d CUDA devices found - reallocating", deviceCount);
    if (deviceCount > 1) {
        dualDevice = 1;
    }



    const unsigned long long oddOffsetOptimization = 2ULL;
    const int dev0 = 0;
    const int dev1 = 1;
    const unsigned long long threadsPerBlock = 256ULL;// 128;// 128; 128=50%, 256=66 on RTX-3500
    unsigned long long cores = 5120ULL;// (argc > 1) ? atoi(argv[1]) : 5120; // get command
    // exited with code -1073741571 any higher
    // VRAM related - cannot exceed 32k threads for dual 12g RTX-3500 - check 4090

    // variables
    // keep these 2 in sync
    unsigned int threadsPower = 14;
    const unsigned long long threads = 16384;
    // diff should be 31 bits (minus oddOffsetOptimization)
    unsigned int startSequencePower = 1;  // do not use 0
    unsigned int endSequencePower = 33;

    // derived
    unsigned long long startSequenceNumber = (1ULL << startSequencePower) + 1ULL;
    unsigned long long endSequenceNumber = (1ULL << endSequencePower) - 1ULL;
    printf("endSequenceNumber: %llu\n", endSequenceNumber);
    // Number of blocks = ceiling(N / threadsPerBlock)
    unsigned int blocks = (threads + threadsPerBlock - 1) / threadsPerBlock;
    size_t size = threads * sizeof(unsigned long long);
    unsigned long long globalMaxValue = startSequenceNumber;
    unsigned long long globalMaxStart = startSequenceNumber;
    unsigned long long iterations = (endSequenceNumber - startSequenceNumber) / oddOffsetOptimization;// +1);
    unsigned long long batchNumberPower = (endSequencePower - startSequencePower) - threadsPower;
    unsigned long long batchNumber = iterations / threads; // 1ULL << batchNumberPower;
    printf("BatchNumberPower: %llu\n", batchNumberPower);
    printf("BatchNumber: %llu\n", batchNumber);
    printf("Iterations: %llu\n", iterations);

    // Host arrays
    unsigned long long host_input0[threads];
    unsigned long long host_result0[threads] = { 0 };
    unsigned long long* device_input0 = nullptr;
    unsigned long long* device_output0 = nullptr;
    unsigned long long host_input1[threads];
    unsigned long long host_result1[threads] = { 0 };
    unsigned long long* device_input1 = nullptr;
    unsigned long long* device_output1 = nullptr;

    time_t timeStart, timeEnd;
    double timeElapsed;
    time(&timeStart);

    // Allocate memory on the GPU
    printf("array allocation bytes per GPU: %d * %d is %d maxSearch: %llu\n", sizeof(unsigned long long), threads, size, endSequenceNumber);
    hipSetDevice(dev0);
    hipMalloc((void**)&device_input0, size);
    hipMalloc((void**)&device_output0, size);
    if (dualDevice > 0) {
        hipSetDevice(dev1);
        hipMalloc((void**)&device_input1, size);
        hipMalloc((void**)&device_output1, size);
    }

    // GPU0: Iterations: 8388608 Threads: 31232 ThreadsPerBlock: 64 Blocks: 488
    printf("GPU0: Threads: %d ThreadsPerBlock: %d Blocks: %d\n", threads, threadsPerBlock, blocks);
    if (dualDevice > 0) {
        printf("GPU1: Threads: %d ThreadsPerBlock: %d Blocks: %d\n", threads, threadsPerBlock, blocks);
    }

    // fill out current batch
    for (int index = 0; index < endSequenceNumber; index++) {
        for (int q = 0; q < threads; q++) {
            host_input0[q] = startSequenceNumber;
            if (dualDevice > 0) {
                startSequenceNumber += oddOffsetOptimization;
                host_input1[q] = startSequenceNumber;
            }
            startSequenceNumber += oddOffsetOptimization;
        }

        hipSetDevice(dev0);
        hipMemcpy(device_input0, host_input0, size, hipMemcpyHostToDevice);

        if (dualDevice > 0) {
            hipSetDevice(dev1);
            hipMemcpy(device_input1, host_input1, size, hipMemcpyHostToDevice);
        }

        // Launch kernel
        hipSetDevice(dev0);
        // kernelName<<<numBlocks, threadsPerBlock>>>(parameters...);
        //collatzCUDAKernel << <blocks, threadsPerBlock >> > (device_input0, device_output0, threads);

        if (dualDevice > 0) {
            hipSetDevice(dev1);
        //    collatzCUDAKernel << <blocks, threadsPerBlock >> > (device_input1, device_output1, threads);
        }

        // Wait for GPU to finish before accessing on host
        hipSetDevice(dev0);
        hipDeviceSynchronize();
        if (dualDevice > 0) {
            hipSetDevice(dev1);
            hipDeviceSynchronize();
        }

        // Copy result from device back to host
        hipMemcpy(host_result0, device_output0, size, hipMemcpyDeviceToHost);
        if (dualDevice > 0) {
            hipMemcpy(host_result1, device_output1, size, hipMemcpyDeviceToHost);
        }

        // parallelize
        for (int index = 0; index < threads; index++) {
            if (host_result0[index] > globalMaxValue) {
                globalMaxValue = host_result0[index];
                globalMaxStart = host_input0[index];
                time(&timeEnd);
                timeElapsed = difftime(timeEnd, timeStart);
                std::cout << "GPU0:Sec: " << timeElapsed << " GlobalMax: " << globalMaxStart << ": " << globalMaxValue << " last search: " << startSequenceNumber << "\n";
            }
            if (dualDevice > 0) {
                if (host_result1[index] > globalMaxValue) {
                    globalMaxValue = host_result1[index];
                    globalMaxStart = host_input1[index];
                    time(&timeEnd);
                    timeElapsed = difftime(timeEnd, timeStart);
                    std::cout << "GPU1:Sec: " << timeElapsed << " GlobalMax: " << globalMaxStart << ": " << globalMaxValue << " last search: " << startSequenceNumber << "\n";
                }
            }
        }
    }

    // Print the result
    std::cout << "collatz:\n";
    for (int i = 0; i < 20/*threads*/; i++)
    {
        std::cout << "GPU0: " << i << ": " << host_input0[i] << " = " << host_result0[i] << "\n";
        if (dualDevice > 0) {
            std::cout << "GPU1: " << i << ": " << host_input1[i] << " = " << host_result1[i] << "\n";
        }
    }

    time(&timeEnd);
    timeElapsed = difftime(timeEnd, timeStart);

    printf("duration: %.f\n", timeElapsed);
    std::cout << "Sec: " << timeElapsed << " GlobalMax: " << globalMaxStart << " : " << globalMaxValue << " last search : " << startSequenceNumber << "\n";

    // Free GPU memory
    hipSetDevice(dev0);
    hipFree(device_input0);
    hipFree(device_output0);
    if (dualDevice > 0) {
        hipSetDevice(dev1);
        hipFree(device_input1);
        hipFree(device_output1);
    }

    free(host_input0);
    if (dualDevice > 0) {
        free(host_input1);
    }
    return;
}

void testCollatzCUDAKernel(unsigned long long _input1, unsigned long long _input0,
    unsigned long long _output1, unsigned long long _output0)//, int threads)
{
    const unsigned long long MAXBIT = 9223372036854775808ULL;
    // Calculate this thread's index
    int threadIndex = 0; //blockDim.x* blockIdx.x + threadIdx.x;

    // Check boundary (in case N is not a multiple of blockDim.x)
    int path = 0;
    unsigned long long max0 = _input0;// [threadIndex] ;
    unsigned long long current0 = _input0;// [threadIndex] ;
    unsigned long long max1 = _input1;// [threadIndex] ;
    unsigned long long current1 = _input1;// [threadIndex] ;
    unsigned long long temp0 = 0ULL;
    unsigned long long temp1 = 0ULL;
    unsigned long long temp0_shift = 0ULL;
    unsigned long long temp0_add = 0ULL;
   

   // if (threadIndex < threads) {
        path = 0;
        max0 = _input0;// [threadIndex] ;
        current0 = _input0;// [threadIndex] ;
        do {
            path += 1;
            // keep copy of n
            //temp0 = current0;
            //temp1 = current1;
            // both even odd include a shift right - but 128 bit 2 bit carry math is required for large numbers at the 64 bit boundary
            // even
            if (current0 % 2ULL == 0) {
                current0 = current0 >> 1;
                // shift high byte if not odd (we already have a 0 in the MSB of the low word - no overflow will occur
                if (current1 % 2ULL != 0) {
                    // add carry to avoid - overflow during the msb add to the low word
                    //temp0_sh = current0;
                    current0 += MAXBIT; // check overflow - will be none
                    //if (current0 < temp0_sh) {
                    //    current1 += 1ULL;
                    //}
                }
                current1 = current1 >> 1;
                printf("even: %llu:%llu\n", current1, current0);
            } else {
                // odd n << 1 + n + 1
                //path += 1; // if we combine odd/even
                // use (n >> 1) + ceil(n) + 1 - only if 128 2 bit carry handling between
                // do only 128-64 bit 3n part of 3n+1 (don't worry about overflow past 128bit into 256 bit space until we get past 64 bit inputs)
                // HIGH (3N)
                if (current0 > MAXBIT) {
                    printf("msb non-zero: %llu:%llu\n", current1, current0);
                }
                current1 *= 3ULL; 

                // LOW (3N + 1) with 2 bit overflow
                // shift first plus carry in (do add n later)
                temp0_shift = (current0 << 1) + 1ULL;
                // if lt - we have overflow
                if (!(current0 < MAXBIT)) {
                    current1 += 1ULL; // add overflow
                }

                // add n step for odd - separate to break out possible 2 bit 64 bit boundary overflow
                temp0_add = temp0_shift + current0;
                if (temp0_add < current0) {
                    current1 += 1ULL; // add overflow
                }
                current0 = temp0_add;
                
                printf("odd:  %llu:%llu\n", current1, current0);
                // check for max (if combined odd/even mult by 2)
                if (max1 < current1) {
                    max1 = current1;
                    max0 = current0;
                    printf("Max1: %llu:%llu\n", current1, current0);
                }
                else {
                    if (max1 == current1) {
                        if (max0 < current0) {
                            max0 = current0;
                            printf("Max0: %llu:%llu\n", current1, current0);
                        }
                    }
                }
            }
        } while (!((current0 == 1ULL) && (current1 == 0ULL)));

        // double max
        //unsigned long long _max1 = 0ULL;
        //unsigned long long _max0 = 0ULL;
        //_max1 = max1 << 1;
        //_max0 = max0 << 1;
        //if (!(max0 < MAXBIT)) {// _max0 < max0) {
        //    _max1 += 1ULL; // add carry
        //}
        printf("path: %llu actual max: %llu:%llu\n", path, max1, max0);// _max1, _max0 );
    //}
    _output0 = max0;
    _output1 = max1;
}

int main(int argc, char* argv[])
{
    int cores = (argc > 1) ? atoi(argv[1]) : 5120; // get command
    singleGPUSearch();
    //dualGPUSearch();
    //unsigned long long _input0 = 12327829503ULL; // 1:2275654840695500112
    unsigned long long _input0 = 23035537407ULL; // 3:13497924420419572192
    //unsigned long long _input0 = 65536ULL;
    unsigned long long _input1 = 0ULL;// 65536ULL;
    unsigned long long _output1 = 0ULL;
    unsigned long long _output0 = 0ULL;
    //testCollatzCUDAKernel(_input1, _input0, _output1, _output0);

    return 0;
}


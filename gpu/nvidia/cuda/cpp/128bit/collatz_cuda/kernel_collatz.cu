#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <time.h>

/**
* Michael O'Brien 20241223
* michael at obrienlabs.dev
* Collatz sequence running on NVidia GPUs like the RTX-3500 ada,A4000,A4500,4090 ada and A6000
* http://www.ericr.nl/wondrous/pathrecs.html
* https://github.com/ObrienlabsDev/performance
* https://github.com/obrienlabs/benchmark/blob/master/ObjectiveC/128bit/main.m
* https://github.com/obrienlabs/benchmark/blob/master/collatz_vs10/collatz_vs10/collatz_vs10.cpp
* https://github.com/ObrienlabsDev/cuda/blob/main/add_example/kernel_collatz.cu
* https://github.com/ObrienlabsDev/collatz/blob/main/src/main/java/dev/obrienlabs/collatz/service/CollatzUnitOfWork.java
* 
*/


/* CUDA Kernel runs on GPU device streaming core */
__global__ void addArrays(unsigned long long* _input, unsigned long long* _output, int threads)//, unsigned long long iterations)
{
    // Calculate this thread's index
    int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Check boundary (in case N is not a multiple of blockDim.x)
    int path = 0;
    unsigned long long max = _input[threadIndex];
    unsigned long long current = _input[threadIndex];

    if (threadIndex < threads)
    {
        //  sec on a mobile RTX-3500 ada 
        //for (unsigned long q = 0; q < iterations; q++) {
            path = 0;
            max = _input[threadIndex];
            current = _input[threadIndex];

            do {
                path += 1;
                if (current % 2 == 0) {
                    current = current >> 1;
                }
                else {
                    current = 1 + current * 3;
                    if (current > max) {
                        max = current;
                    }
                }
            } while (current > 1);
        //}
    }
    _output[threadIndex] = max;
}

/* Host progrem */
int main(int argc, char* argv[])
{
    int deviceCount = 0;
    int dualDevice = 0;
    hipGetDeviceCount(&deviceCount);
    printf("%d devices found - reallocating", deviceCount);
    if (deviceCount > 1) {
        dualDevice = 1;
    }

    const int dev0 = 0;
    const int dev1 = 1;

    int cores = (argc > 1) ? atoi(argv[1]) : 5120; // get command
    // exited with code -1073741571 any higher
    const int threads = 32768 - 1536;// 7168 * 4;
    // GPU0: Iterations: 8388608 Threads : 31232 ThreadsPerBlock : 64 Blocks : 488
    //int iterationPower = 17;// 23;
    //unsigned long long iterations = 1 << iterationPower;
    const int threadsPerBlock = 128;

    // debug is 32x slower than release
    // iterpower,threadsPerBlock,cores,seconds
    // RTX-3500 Ada
    // 256 threads per block is double the SM core count of 128 cores per SM:
    // 22, 256, 4096 = 130s
    // 22, 128, 4096 = 124
    // 22, 256, 5120 = 132
    // 22, 128, 5120 = 125
    // 22, 64. 5120  = 125

    // 4090
    // 22,64,5120, 94, 25 TDP
    // 22,128,5120, 94
    // 22,256,5120, 99
    // 22,128,16384, 99, 35 TDP
    // 22,128,16384, 94, 35 TDP exe
    // 23,128, 7168x8,128,229
    // 
    // RTX-a4500 Ampere
    // 22,64,5120, 140 exe 53 TDP


    // Host arrays
    unsigned long long host_input0[threads];
    unsigned long long host_input1[threads];

    unsigned long long startSequence = 1L;
    for (int q = 0; q < threads; q++) {
        host_input0[q] = startSequence;// 8528817511;
        if (dualDevice > 0) {
            startSequence += 2;
            host_input1[q] = startSequence;// 8528817511;
        }
        startSequence += 2;
    }

    

    unsigned long long host_result0[threads] = { 0 };
    unsigned long long host_result1[threads] = { 0 };

    // Device pointers
    unsigned long long* device_input0 = nullptr;
    unsigned long long* device_output0 = nullptr;
    unsigned long long* device_input1 = nullptr;
    unsigned long long* device_output1 = nullptr;

    time_t timeStart, timeEnd;
    double timeElapsed;

    time(&timeStart);

    //int N_per_gpu = N / 2;
    // Allocate memory on the GPU
    size_t size = threads * sizeof(unsigned long long);
    printf("array allocation bytes per GPU: %d * %d is %d maxSearch: %lld\n", sizeof(unsigned long long), threads, size, startSequence);
    // Number of blocks = ceiling(N / threadsPerBlock)
    int blocks = (threads + threadsPerBlock - 1) / threadsPerBlock;

    hipSetDevice(dev0);
    hipMalloc((void**)&device_input0, size);
    hipMalloc((void**)&device_output0, size);
    // Copy input data from host to device
    hipMemcpy(device_input0, host_input0, size, hipMemcpyHostToDevice);

    if (dualDevice > 0) {
        hipSetDevice(dev1);
        hipMalloc((void**)&device_input1, size);
        hipMalloc((void**)&device_output1, size);
        // Copy input data from host to device
        hipMemcpy(device_input1, host_input1, size, hipMemcpyHostToDevice);
    }

    // maximums for 4090 single 2*28672 or split - 4.7A
    // 32k - 1.5k
    // GPU0: Iterations: 8388608 Threads: 31232 ThreadsPerBlock: 64 Blocks: 488
    printf("GPU0: Threads: %d ThreadsPerBlock: %d Blocks: %d\n", threads, threadsPerBlock, blocks);
 
    // Launch kernel
    hipSetDevice(dev0);
    // kernelName<<<numBlocks, threadsPerBlock>>>(parameters...);
    addArrays << <blocks, threadsPerBlock >> > (device_input0, device_output0, threads);

    if (dualDevice > 0) {
        printf("GPU1: Threads: %d ThreadsPerBlock: %d Blocks: %d\n", threads, threadsPerBlock, blocks);
        hipSetDevice(dev1);
        // kernelName<<<numBlocks, threadsPerBlock>>>(parameters...);
        addArrays << <blocks, threadsPerBlock >> > (device_input1, device_output1, threads);
    }

    // Wait for GPU to finish before accessing on host
    hipSetDevice(dev0);
    hipDeviceSynchronize();
    if (dualDevice > 0) {
        hipSetDevice(dev1);
        hipDeviceSynchronize();
    }

    // Copy result from device back to host
    hipMemcpy(host_result0, device_output0, size, hipMemcpyDeviceToHost);
    if (dualDevice > 0) {
        hipMemcpy(host_result1, device_output1, size, hipMemcpyDeviceToHost);
    }

    // Print the result
    std::cout << "collatz:\n";
    int i = 0;
    for (int i = 0; i < 20/*threads*/; i++)
    {
        std::cout << "GPU0: " << i << ": " << host_input0[i] << " = " << host_result0[i] << "\n";
        if (dualDevice > 0) {
            std::cout << "GPU1: " << i << ": " << host_input1[i] << " = " << host_result1[i] << "\n";
        }
    }

    time(&timeEnd);
    timeElapsed = difftime(timeEnd, timeStart);

    //std::cout << "2 + 7 = " << c << std::endl;
    printf("duration: %.f\n", timeElapsed);

    // Free GPU memory
    hipSetDevice(dev0);
    hipFree(device_input0);
    hipFree(device_output0);
    if (dualDevice > 0) {
        hipSetDevice(dev1);
        hipFree(device_input1);
        hipFree(device_output1);
    }

    free(host_input0);
    if (dualDevice > 0) {
        free(host_input1);
    }

    return 0;
}


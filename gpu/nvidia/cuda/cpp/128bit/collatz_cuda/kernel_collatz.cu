#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <time.h>

/**
* Michael O'Brien 20241223
* michael at obrienlabs.dev
* 128 bit version
* Collatz sequence running on NVidia GPUs like the RTX-3500 ada,A4000,A4500,4090 ada and A6000
* http://www.ericr.nl/wondrous/pathrecs.html
* https://github.com/ObrienlabsDev/performance
* https://github.com/obrienlabs/benchmark/blob/master/ObjectiveC/128bit/main.m
* https://github.com/obrienlabs/benchmark/blob/master/collatz_vs10/collatz_vs10/collatz_vs10.cpp
* https://github.com/ObrienlabsDev/cuda/blob/main/add_example/kernel_collatz.cu
* https://github.com/ObrienlabsDev/collatz/blob/main/src/main/java/dev/obrienlabs/collatz/service/CollatzUnitOfWork.java
*/

__global__ void collatzCUDAKernel(/*unsigned long long* _input1, */unsigned long long* _input0,
    unsigned long long* _output1, unsigned long long* _output0, int threads)
{
    const unsigned long long MAXBIT = 9223372036854775808;
    // Calculate this thread's index
    int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Check boundary (in case N is not a multiple of blockDim.x)
    int path = 0;
    unsigned long long max0 = _input0[threadIndex];
    unsigned long long current0 = _input0[threadIndex];
    unsigned long long max1 = 0ULL;// _input1[threadIndex];
    unsigned long long current1 = 0ULL; //_input1[threadIndex];
    unsigned long long temp1 = 0ULL;
    unsigned long long temp0_sh = 0ULL;
    unsigned long long temp0_ad = 0ULL;

    if (threadIndex < threads) {
            path = 0;
            max0 = _input0[threadIndex];
            current0 = _input0[threadIndex];
            do {
                path += 1;
                if (current0 % 2 == 0) {
                    current0 = current0 >> 1;
                    // shift high byte if not odd
                    if (current1 % 2 != 0) {
                        current0 += MAXBIT;
                    }
                    else {
                        current1 = current1 >> 1;
                    }
                }
                else {
                    temp1 = 3 * current1;// + (current1 << 1);
                    current1 = temp1;

                    // shift first - calc overflow 1
                    temp0_sh = 1 + (current0 << 1);
                    if (!(current0 < MAXBIT)) {
                        current1 = current1 + 1;
                    }
                    // add second - calc overflow 2
                    temp0_ad = temp0_sh + current0;
                    if (temp0_ad < current0) { // overflow
                        current1 = current1 + 1;
                    }
                    current0 = temp0_ad;

                    // check for max
                    if (max1 < current1) {
                        max1 = current1;
                        max0 = current0;
                    }
                    else {
                        if (max1 == current1) {
                            if (max0 < current0) {
                                max0 = current0;
                            }
                        }
                    }
                }
            } while (!(current0 == 1) && (current1 == 0));
    }
    _output0[threadIndex] = max0;
}

void singleGPUSearch() {
    unsigned long long MAXBIT = 9223372036854775808;
    int deviceCount = 0;
    int dualDevice = 0;
    hipGetDeviceCount(&deviceCount);
    printf("%d CUDA devices found - reallocating\n", deviceCount);
    if (deviceCount > 1) {
        dualDevice = 1;
    }

    const unsigned long long oddOffsetOptimization = 2ULL;
    const int dev0 = 0;
    const unsigned long long threadsPerBlock = 256ULL;// 128;// 128; 128=50%, 256=66 on RTX-3500
    unsigned long long cores = 5120ULL;// (argc > 1) ? atoi(argv[1]) : 5120; // get command

    // variables
    // keep these 2 in sync
    unsigned int threadsPower = 15;
    const unsigned long long threads = 32768;
    // diff should be 31 bits (minus oddOffsetOptimization)
    unsigned int startSequencePower = 1;  // do not use 0
    unsigned int endSequencePower = 32; 

    // derived
    unsigned long long startSequenceNumber = (1ULL << startSequencePower) + 1ULL;
    unsigned long long endSequenceNumber = (1ULL << endSequencePower) - 1ULL;
    printf("endSequenceNumber: %llu\n", endSequenceNumber);
    // Number of blocks = ceiling(N / threadsPerBlock)
    unsigned int blocks = (threads + threadsPerBlock - 1) / threadsPerBlock;
    size_t size = threads * sizeof(unsigned long long);
    unsigned long long globalMaxValue0 = startSequenceNumber;
    unsigned long long globalMaxStart0 = startSequenceNumber;
    unsigned long long globalMaxValue1 = 0ULL;
    unsigned long long globalMaxStart1 = 0ULL;
    unsigned long long iterations = (endSequenceNumber - startSequenceNumber) / oddOffsetOptimization;
    unsigned long long batchNumberPower = (endSequencePower - startSequencePower) - threadsPower;
    unsigned long long batchNumber = iterations / threads; // 1ULL << batchNumberPower;
    printf("BatchNumberPower: %llu\n", batchNumberPower);
    printf("BatchNumber: %llu\n", batchNumber);
    printf("Iterations: %llu\n", iterations);

    // Host arrays
    unsigned long long host_input0[threads];
    //unsigned long long host_input1[threads];
    unsigned long long host_result0[threads] = { 0 };
    unsigned long long* device_input0 = nullptr;
    unsigned long long* device_output0 = nullptr;
    // for 128 not 2nd GPU
    //unsigned long long* device_input1 = nullptr;
    unsigned long long* device_output1 = nullptr;
    unsigned long long host_result1[threads] = { 0 };

    time_t timeStart, timeEnd;
    double timeElapsed;
    time(&timeStart);

    // Allocate memory on the GPU
    printf("array allocation bytes per GPU: %d * %d is %d maxSearch: %llu to %llu\n", sizeof(unsigned long long) * 2, threads, size, startSequenceNumber, endSequenceNumber);
    hipSetDevice(dev0);
    hipMalloc((void**)&device_input0, size);
    //hipMalloc((void**)&device_input1, size);
    hipMalloc((void**)&device_output0, size);
    hipMalloc((void**)&device_output1, size);

    // Iterations = 2 ^ (15(threads) + 16(endSequence = runs) + 1(odd multiplier))
    printf("GPU0: Iterations: %llu via (Threads: %llu * Batches: %d * 2 (odd mult)) ThreadsPerBlock: %d Blocks: %d\n", 
        iterations, threads, batchNumber, threadsPerBlock, blocks);
    for (int batch = 0; batch < batchNumber; batch++) {
        // prepare inputs
        for (int thread = 0; thread < threads; thread++) {
            host_input0[thread] = startSequenceNumber;
            //host_input1[thread] = 0ULL;
            startSequenceNumber += oddOffsetOptimization;
        }

        hipMemcpy(device_input0, host_input0, size, hipMemcpyHostToDevice);
        //hipMemcpy(device_input1, host_input1, size, hipMemcpyHostToDevice);
        // Launch kernel
        // kernelName<<<numBlocks, threadsPerBlock>>>(parameters...);
        collatzCUDAKernel << <blocks, threadsPerBlock >> > (/*device_input1, */device_input0, device_output1, device_output0, threads);

        // Wait for GPU to finish before accessing on host
        hipDeviceSynchronize();

        // Copy result from device back to host
        hipMemcpy(host_result0, device_output0, size, hipMemcpyDeviceToHost);
        hipMemcpy(host_result1, device_output1, size, hipMemcpyDeviceToHost);

        // process reesults: parallelize with OpenMP
        for (int thread = 0; thread < threads; thread++) {
            if (host_result1[thread] > globalMaxValue1) {
                globalMaxValue0 = host_result0[thread];
                globalMaxValue1 = host_result1[thread];
                globalMaxStart0 = host_input0[thread];
                globalMaxStart1 = 0ULL;// host_input1[thread];

                time(&timeEnd);
                timeElapsed = difftime(timeEnd, timeStart);
                std::cout << "GPU0:Sec: " << timeElapsed << " GlobalMax: " << globalMaxStart1 << ":" << globalMaxStart0 << ": " << globalMaxValue1 
                    << ":" <<globalMaxValue0 << " last search: " << startSequenceNumber << "\n";
            } else {
                // handle only lsb gt
                if (host_result1[thread] == globalMaxValue1) {
                    if(host_result0[thread] > globalMaxValue0) {
                        globalMaxValue0 = host_result0[thread];
                        time(&timeEnd);
                        timeElapsed = difftime(timeEnd, timeStart);
                        std::cout << "GPU0:Sec: " << timeElapsed << " GlobalMax: " << globalMaxStart1 << ":" << globalMaxStart0 << ": " << globalMaxValue1
                            << ":" << globalMaxValue0 << " last search: " << startSequenceNumber << "\n";
                    }
                }
            }
            // TODO: maxPath
        }
    }

    // Print the result for the last run
    std::cout << "collatz:\n";
    for (int i = 0; i < 20/*threads*/; i++)
    {
        std::cout << "GPU0: " << i << ": " << /*host_input1[i] <<*/ ":" << host_input0[i] << " = " << host_result1[i] << host_result0[i] << "\n";
    }

    time(&timeEnd);
    timeElapsed = difftime(timeEnd, timeStart);

    printf("duration: %.f\n", timeElapsed);
    std::cout << "Sec: " << timeElapsed << " GlobalMax: " << globalMaxStart1 << ":" << globalMaxStart0 << " : " << globalMaxValue1 
        << ":" << globalMaxValue0 << " last search : " << startSequenceNumber << "\n";

    // Free GPU memory
    hipFree(device_input0);
    //hipFree(device_input1);
    hipFree(device_output0);
    hipFree(device_output1);

    free(host_input0);
    //free(host_input1);
    free(host_result0);
    free(host_result1);
    return;
}

void dualGPUSearch() {
    int deviceCount = 0;
    int dualDevice = 0;
    hipGetDeviceCount(&deviceCount);
    printf("%d CUDA devices found - reallocating", deviceCount);
    if (deviceCount > 1) {
        dualDevice = 1;
    }



    const unsigned long long oddOffsetOptimization = 2ULL;
    const int dev0 = 0;
    const int dev1 = 1;
    const unsigned long long threadsPerBlock = 256ULL;// 128;// 128; 128=50%, 256=66 on RTX-3500
    unsigned long long cores = 5120ULL;// (argc > 1) ? atoi(argv[1]) : 5120; // get command
    // exited with code -1073741571 any higher
    // VRAM related - cannot exceed 32k threads for dual 12g RTX-3500 - check 4090

    // variables
    // keep these 2 in sync
    unsigned int threadsPower = 14;
    const unsigned long long threads = 16384;
    // diff should be 31 bits (minus oddOffsetOptimization)
    unsigned int startSequencePower = 1;  // do not use 0
    unsigned int endSequencePower = 32;

    // derived
    unsigned long long startSequenceNumber = (1ULL << startSequencePower) + 1ULL;
    unsigned long long endSequenceNumber = (1ULL << endSequencePower) - 1ULL;
    printf("endSequenceNumber: %llu\n", endSequenceNumber);
    // Number of blocks = ceiling(N / threadsPerBlock)
    unsigned int blocks = (threads + threadsPerBlock - 1) / threadsPerBlock;
    size_t size = threads * sizeof(unsigned long long);
    unsigned long long globalMaxValue = startSequenceNumber;
    unsigned long long globalMaxStart = startSequenceNumber;
    unsigned long long iterations = (endSequenceNumber - startSequenceNumber) / oddOffsetOptimization;// +1);
    unsigned long long batchNumberPower = (endSequencePower - startSequencePower) - threadsPower;
    unsigned long long batchNumber = iterations / threads; // 1ULL << batchNumberPower;
    printf("BatchNumberPower: %llu\n", batchNumberPower);
    printf("BatchNumber: %llu\n", batchNumber);
    printf("Iterations: %llu\n", iterations);

    // Host arrays
    unsigned long long host_input0[threads];
    unsigned long long host_result0[threads] = { 0 };
    unsigned long long* device_input0 = nullptr;
    unsigned long long* device_output0 = nullptr;
    unsigned long long host_input1[threads];
    unsigned long long host_result1[threads] = { 0 };
    unsigned long long* device_input1 = nullptr;
    unsigned long long* device_output1 = nullptr;

    time_t timeStart, timeEnd;
    double timeElapsed;
    time(&timeStart);

    // Allocate memory on the GPU
    printf("array allocation bytes per GPU: %d * %d is %d maxSearch: %lld\n", sizeof(unsigned long long), threads, size, endSequenceNumber);
    hipSetDevice(dev0);
    hipMalloc((void**)&device_input0, size);
    hipMalloc((void**)&device_output0, size);
    if (dualDevice > 0) {
        hipSetDevice(dev1);
        hipMalloc((void**)&device_input1, size);
        hipMalloc((void**)&device_output1, size);
    }

    // GPU0: Iterations: 8388608 Threads: 31232 ThreadsPerBlock: 64 Blocks: 488
    printf("GPU0: Threads: %d ThreadsPerBlock: %d Blocks: %d\n", threads, threadsPerBlock, blocks);
    if (dualDevice > 0) {
        printf("GPU1: Threads: %d ThreadsPerBlock: %d Blocks: %d\n", threads, threadsPerBlock, blocks);
    }

    // fill out current batch
    for (int index = 0; index < endSequenceNumber; index++) {
        for (int q = 0; q < threads; q++) {
            host_input0[q] = startSequenceNumber;
            if (dualDevice > 0) {
                startSequenceNumber += oddOffsetOptimization;
                host_input1[q] = startSequenceNumber;
            }
            startSequenceNumber += oddOffsetOptimization;
        }

        hipSetDevice(dev0);
        hipMemcpy(device_input0, host_input0, size, hipMemcpyHostToDevice);

        if (dualDevice > 0) {
            hipSetDevice(dev1);
            hipMemcpy(device_input1, host_input1, size, hipMemcpyHostToDevice);
        }

        // Launch kernel
        hipSetDevice(dev0);
        // kernelName<<<numBlocks, threadsPerBlock>>>(parameters...);
        //collatzCUDAKernel << <blocks, threadsPerBlock >> > (device_input0, device_output0, threads);

        if (dualDevice > 0) {
            hipSetDevice(dev1);
        //    collatzCUDAKernel << <blocks, threadsPerBlock >> > (device_input1, device_output1, threads);
        }

        // Wait for GPU to finish before accessing on host
        hipSetDevice(dev0);
        hipDeviceSynchronize();
        if (dualDevice > 0) {
            hipSetDevice(dev1);
            hipDeviceSynchronize();
        }

        // Copy result from device back to host
        hipMemcpy(host_result0, device_output0, size, hipMemcpyDeviceToHost);
        if (dualDevice > 0) {
            hipMemcpy(host_result1, device_output1, size, hipMemcpyDeviceToHost);
        }

        // parallelize
        for (int index = 0; index < threads; index++) {
            if (host_result0[index] > globalMaxValue) {
                globalMaxValue = host_result0[index];
                globalMaxStart = host_input0[index];
                time(&timeEnd);
                timeElapsed = difftime(timeEnd, timeStart);
                std::cout << "GPU0:Sec: " << timeElapsed << " GlobalMax: " << globalMaxStart << ": " << globalMaxValue << " last search: " << startSequenceNumber << "\n";
            }
            if (dualDevice > 0) {
                if (host_result1[index] > globalMaxValue) {
                    globalMaxValue = host_result1[index];
                    globalMaxStart = host_input1[index];
                    time(&timeEnd);
                    timeElapsed = difftime(timeEnd, timeStart);
                    std::cout << "GPU1:Sec: " << timeElapsed << " GlobalMax: " << globalMaxStart << ": " << globalMaxValue << " last search: " << startSequenceNumber << "\n";
                }
            }
        }
    }

    // Print the result
    std::cout << "collatz:\n";
    for (int i = 0; i < 20/*threads*/; i++)
    {
        std::cout << "GPU0: " << i << ": " << host_input0[i] << " = " << host_result0[i] << "\n";
        if (dualDevice > 0) {
            std::cout << "GPU1: " << i << ": " << host_input1[i] << " = " << host_result1[i] << "\n";
        }
    }

    time(&timeEnd);
    timeElapsed = difftime(timeEnd, timeStart);

    printf("duration: %.f\n", timeElapsed);
    std::cout << "Sec: " << timeElapsed << " GlobalMax: " << globalMaxStart << " : " << globalMaxValue << " last search : " << startSequenceNumber << "\n";

    // Free GPU memory
    hipSetDevice(dev0);
    hipFree(device_input0);
    hipFree(device_output0);
    if (dualDevice > 0) {
        hipSetDevice(dev1);
        hipFree(device_input1);
        hipFree(device_output1);
    }

    free(host_input0);
    if (dualDevice > 0) {
        free(host_input1);
    }
    return;
}

int main(int argc, char* argv[])
{
    int cores = (argc > 1) ? atoi(argv[1]) : 5120; // get command
    singleGPUSearch();
    //dualGPUSearch();
    return 0;
}

